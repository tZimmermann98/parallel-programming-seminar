#include "hip/hip_runtime.h"
template <typename T, typename O, typename F>
__global__ void map_kernel(T* input, O* output, int size, F func){
    // Calculate the global index for the current thread across all blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if index is less than the size of the input vector, apply the function to the input and store the result in the output
    if (idx < size){
        output[idx] = func(input[idx]);
    }
}

template <typename T, typename O, typename F>
__global__ void reduce_kernel(T* input, O* output, int size, F func){
    // Shared memory for each block
    __shared__ O sdata[1024];
    // Get the thread ID within the current block
    unsigned int tid = threadIdx.x;
    // Calculate the global index for the current thread across all blocks
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size){
        // if index is less than the size of the input vector, store the input in the shared memory
        sdata[tid] = input[idx];
    }
    else {
        // else store 0 in the shared memory
        sdata[tid] = 0;
    }
    // Wait for all threads to finish writing to shared memory
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
        // if the thread ID is less than the current shared memory size
        if (tid < s){
            // apply the function to the value at current index and at index + the current shared memory size
            sdata[tid] = func(sdata[tid], sdata[tid + s]);
        }
        // Wait for all threads to finish writing to shared memory
        __syncthreads();
    }

    if (tid == 0){
        // Store the result of the reduction in the output vector
        output[blockIdx.x] = sdata[0];
    }
    
}

template <typename T1, typename T2, typename O, typename F>
__global__ void zip_kernel(T1* input1, T2* input2, O* output, int size, F func){
    // Calculate the global index for the current thread across all blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        // if index is less than the size of the input vectors, apply the function to the inputs and store the result in the output
        output[idx] = func(input1[idx], input2[idx]);
    }
}

template<typename T, typename F>
void map(std::vector <T>& input, std::vector <T>& output, F func, int numThreads, float& map_copy_device, float& map_kernel_time, float& map_copy_host, float& map_total){
    // Start overall timer
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    // Get the size of the input vector and initialize device pointers
    int size = input.size();
    T* d_input;
    T* d_output;

    // Allocate memory on the device
    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, size * sizeof(T));
    
    // Start timer for copying input vector to device
    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    // Copy input vector to device
    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);

    // Stop timer for copying input vector to device
    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&map_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    // inititalize block and grid dimensions
    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    // Start timer for kernel
    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    // Call the map kernel
    map_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, size, func);

    // Stop timer for kernel
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&map_kernel_time, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    // Start timer for copying output vector to host
    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    // Copy output vector to host
    hipMemcpy(output.data(), d_output, size * sizeof(T), hipMemcpyDeviceToHost);

    // Stop timer for copying output vector to host
    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&map_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Stop overall timer
    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&map_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}

template<typename T, typename F>
void reduce(std::vector <T>& input, T& output, F func, int numThreads, float& reduce_copy_device, float& reduce_kernel_time, float& reduce_copy_host, float& reduce_total){
    // Start overall timer
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    // Get the size of the input vector and initialize device pointers
    int size = input.size();
    T* d_input;
    T* d_output;
    T* d_final_output;

    // Allocate memory on the device
    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, sizeof(T) * ((size + 1023) / 1024));
    hipMalloc(&d_final_output, sizeof(T));

    // Start timer for copying input vector to device
    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    // Copy input vector to device
    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);

    // Stop timer for copying input vector to device
    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&reduce_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    // inititalize block and grid dimensions
    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    // Start timer for kernel
    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    // Call the reduce kernel for intermediate results
    reduce_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, size, func);

    // Call the reduce kernel for final result
    reduce_kernel<<<1, dimGrid.x>>>(d_output, d_final_output, dimGrid.x, func);

    // Stop timer for kernel
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&reduce_kernel_time, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    // Start timer for copying output vector to host
    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    // Copy output vector to host
    hipMemcpy(&output, d_final_output, sizeof(T), hipMemcpyDeviceToHost);

    // Stop timer for copying output vector to host
    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&reduce_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_final_output);

    // Stop overall timer
    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&reduce_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}

template<typename T1, typename T2, typename T3, typename F>
void zip(std::vector <T1>& input1, std::vector <T2>& input2, std::vector <T3>& output, F func, int numThreads, float& zip_copy_device, float& zip_kernel_time, float& zip_copy_host, float& zip_total){
    // Start overall timer
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    // Get the size of the input vector and initialize device pointers
    int size = std::min(input1.size(), input2.size());
    T1* d_input1;
    T2* d_input2;
    T3* d_output;

    // Allocate memory on the device
    hipMalloc(&d_input1, size * sizeof(T1));
    hipMalloc(&d_input2, size * sizeof(T2));
    hipMalloc(&d_output, size * sizeof(T3));

    // Start timer for copying input vectors to device
    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    // Copy input vectors to device
    hipMemcpy(d_input1, input1.data(), size * sizeof(T1), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2.data(), size * sizeof(T2), hipMemcpyHostToDevice);

    // Stop timer for copying input vectors to device
    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&zip_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    // inititalize block and grid dimensions
    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    // Start timer for kernel
    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    // Call the zip kernel
    zip_kernel<<<dimGrid, dimBlock>>>(d_input1, d_input2, d_output, size, func);

    // Stop timer for kernel
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&zip_kernel_time, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    // Start timer for copying output vector to host
    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    // Copy output vector to host
    hipMemcpy(output.data(), d_output, size * sizeof(T3), hipMemcpyDeviceToHost);

    // Stop timer for copying output vector to host
    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&zip_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    // Free device memory
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);

    // Stop overall timer
    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&zip_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}