#include "hip/hip_runtime.h"
template <typename T, typename O, typename F>
__global__ void map_kernel(T* input, O* output, int size, F func){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        output[idx] = func(input[idx]);
    }
}

template <typename T, typename O, typename F>
__global__ void reduce_kernel(T* input, O* output, int size, F func){
    __shared__ O sdata[1024];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        sdata[tid] = input[i];
    }
    else {
        sdata[tid] = 0;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
        if (tid < s){
            sdata[tid] = func(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0){
        output[blockIdx.x] = sdata[0];
    }
    
}

template <typename T1, typename T2, typename O, typename F>
__global__ void zip_kernel(T1* input1, T2* input2, O* output, int size, F func){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        output[idx] = func(input1[idx], input2[idx]);
    }
}

template<typename T, typename F>
void map(std::vector <T>& input, std::vector <T>& output, F func, int numThreads, float& map_copy_device, float& map_kernel, float& map_copy_host, float& map_total){
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    int size = input.size();
    T* d_input;
    T* d_output;

    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, size * sizeof(T));
    
    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);

    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&map_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    map_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, size, func);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&map_kernel, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    hipMemcpy(output.data(), d_output, size * sizeof(T), hipMemcpyDeviceToHost);

    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&map_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    hipFree(d_input);
    hipFree(d_output);

    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&map_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}

template<typename T, typename F>
void reduce(std::vector <T>& input, T& output, F func, int numThreads, float& reduce_copy_device, float& reduce_kernel, float& reduce_copy_host, float& reduce_total){
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    int size = input.size();

    T* d_input;
    T* d_output;
    T* d_final_output;

    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, sizeof(T) * ((size + 1023) / 1024));
    hipMalloc(&d_final_output, sizeof(T));

    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);

    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&reduce_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    reduce_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, size, func);

    reduce_kernel<<<1, dimGrid.x>>>(d_output, d_final_output, dimGrid.x, func);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&reduce_kernel, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    hipMemcpy(&output, d_final_output, sizeof(T), hipMemcpyDeviceToHost);

    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&reduce_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_final_output);

    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&reduce_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}

template<typename T1, typename T2, typename T3, typename F>
void zip(std::vector <T1>& input1, std::vector <T2>& input2, std::vector <T3>& output, F func, int numThreads, float& zip_copy_device, float& zip_kernel, float& zip_copy_host, float& zip_total){
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    int size = input1.size();

    T1* d_input1;
    T2* d_input2;
    T3* d_output;

    hipMalloc(&d_input1, size * sizeof(T1));
    hipMalloc(&d_input2, size * sizeof(T2));
    hipMalloc(&d_output, size * sizeof(T3));

    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    hipMemcpy(d_input1, input1.data(), size * sizeof(T1), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2.data(), size * sizeof(T2), hipMemcpyHostToDevice);

    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&zip_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    zip_kernel<<<dimGrid, dimBlock>>>(d_input1, d_input2, d_output, size, func);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&zip_kernel, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    hipMemcpy(output.data(), d_output, size * sizeof(T3), hipMemcpyDeviceToHost);

    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&zip_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);

    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&zip_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}