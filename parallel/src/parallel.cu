#include "hip/hip_runtime.h"
template <typename T, typename O, typename F>
__global__ void map_kernel(T* input, O* output, int size, F func){
    // Calculate the global index for the current thread across all blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if index is less than the size of the input vector, apply the function to the input and store the result in the output
    if (idx < size){
        output[idx] = func(input[idx]);
    }
}

template <typename T, typename O, typename F>
__global__ void reduce_kernel(T* input, O* output, int size, F func){
    // Shared memory for each block
    __shared__ O sdata[1024];
    // Get the thread ID within the current block
    unsigned int tid = threadIdx.x;
    // Calculate the global index for the current thread across all blocks
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size){
        // if index is less than the size of the input vector, store the input in the shared memory
        sdata[tid] = input[idx];
    }
    else {
        // else store 0 in the shared memory
        sdata[tid] = 0;
    }
    // Wait for all threads to finish writing to shared memory
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
        // if the thread ID is less than the current shared memory size
        if (tid < s){
            // apply the function to the value at current index and at index + the current shared memory size
            sdata[tid] = func(sdata[tid], sdata[tid + s]);
        }
        // Wait for all threads to finish writing to shared memory
        __syncthreads();
    }

    if (tid == 0){
        // Store the result of the reduction in the output vector
        output[blockIdx.x] = sdata[0];
    }
    
}

template <typename T1, typename T2, typename O, typename F>
__global__ void zip_kernel(T1* input1, T2* input2, O* output, int size, F func){
    // Calculate the global index for the current thread across all blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        // if index is less than the size of the input vectors, apply the function to the inputs and store the result in the output
        output[idx] = func(input1[idx], input2[idx]);
    }
}

template<typename T, typename F>
void map(std::vector <T>& input, std::vector <T>& output, F func, int numThreads, float& map_copy_device, float& map_kernel_time, float& map_copy_host, float& map_total){
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    int size = input.size();
    T* d_input;
    T* d_output;

    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, size * sizeof(T));
    
    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);

    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&map_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    map_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, size, func);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&map_kernel_time, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    hipMemcpy(output.data(), d_output, size * sizeof(T), hipMemcpyDeviceToHost);

    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&map_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    hipFree(d_input);
    hipFree(d_output);

    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&map_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}

template<typename T, typename F>
void reduce(std::vector <T>& input, T& output, F func, int numThreads, float& reduce_copy_device, float& reduce_kernel_time, float& reduce_copy_host, float& reduce_total){
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    int size = input.size();

    T* d_input;
    T* d_output;
    T* d_final_output;

    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, sizeof(T) * ((size + 1023) / 1024));
    hipMalloc(&d_final_output, sizeof(T));

    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);

    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&reduce_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    reduce_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, size, func);

    reduce_kernel<<<1, dimGrid.x>>>(d_output, d_final_output, dimGrid.x, func);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&reduce_kernel_time, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    hipMemcpy(&output, d_final_output, sizeof(T), hipMemcpyDeviceToHost);

    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&reduce_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_final_output);

    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&reduce_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}

template<typename T1, typename T2, typename T3, typename F>
void zip(std::vector <T1>& input1, std::vector <T2>& input2, std::vector <T3>& output, F func, int numThreads, float& zip_copy_device, float& zip_kernel_time, float& zip_copy_host, float& zip_total){
    hipEvent_t start_all, stop_all;
    hipEventCreate(&start_all);
    hipEventCreate(&stop_all);
    hipEventRecord(start_all);

    int size = std::min(input1.size(), input2.size());

    T1* d_input1;
    T2* d_input2;
    T3* d_output;

    hipMalloc(&d_input1, size * sizeof(T1));
    hipMalloc(&d_input2, size * sizeof(T2));
    hipMalloc(&d_output, size * sizeof(T3));

    hipEvent_t start_copy_device, stop_copy_device;
    hipEventCreate(&start_copy_device);
    hipEventCreate(&stop_copy_device);
    hipEventRecord(start_copy_device);

    hipMemcpy(d_input1, input1.data(), size * sizeof(T1), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2.data(), size * sizeof(T2), hipMemcpyHostToDevice);

    hipEventRecord(stop_copy_device);
    hipEventSynchronize(stop_copy_device);
    hipEventElapsedTime(&zip_copy_device, start_copy_device, stop_copy_device);
    hipEventDestroy(start_copy_device);
    hipEventDestroy(stop_copy_device);

    dim3 dimBlock(numThreads);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventRecord(start_kernel);

    zip_kernel<<<dimGrid, dimBlock>>>(d_input1, d_input2, d_output, size, func);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&zip_kernel_time, start_kernel, stop_kernel);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);

    hipEvent_t start_copy_host, stop_copy_host;
    hipEventCreate(&start_copy_host);
    hipEventCreate(&stop_copy_host);
    hipEventRecord(start_copy_host);

    hipMemcpy(output.data(), d_output, size * sizeof(T3), hipMemcpyDeviceToHost);

    hipEventRecord(stop_copy_host);
    hipEventSynchronize(stop_copy_host);
    hipEventElapsedTime(&zip_copy_host, start_copy_host, stop_copy_host);
    hipEventDestroy(start_copy_host);
    hipEventDestroy(stop_copy_host);

    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);

    hipEventRecord(stop_all);
    hipEventSynchronize(stop_all);
    hipEventElapsedTime(&zip_total, start_all, stop_all);
    hipEventDestroy(start_all);
    hipEventDestroy(stop_all);
}